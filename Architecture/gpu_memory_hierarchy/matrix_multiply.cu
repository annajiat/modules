/*
 * CUDA program to multiply matrices (fills in matrices itself)
 * 
 * compile with:
 *      nvcc -o matrix_multiply matrix_multiply.cu
 *
 * run with:
 *      ./matrix_multiply
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <cstdlib>

//constants to control the program:
#define NTESTS 1           /* # of tests to run */
#define TILE_WIDTH 32      /* # of threads in each dimension per block */
                           /* #threads per block = TILE_WIDTH * TILE_WIDTH */
#define WIDTH 1024         /* matrix dimensions (assumes square matrix) */


__global__ void kernel(float* Md, float* Nd, float* Pd, int width) {
  //method to run on GPU; called once per element of output matrix
  
  //calculate indices for the element to compute:
  int row = blockIdx.y*TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x*TILE_WIDTH + threadIdx.x;

  if(row >= width || col >= width)  //check that indices are in bounds
    return;  

  float tmp = 0;  //local variable in which to accumulate the answer
  for(int k=0; k < width; ++k)
    tmp += Md[row*width + k] * Nd[k*width+col];
  Pd[row*width+col] =  tmp;
}

void verify_solution(float *a, float *b, float *c, int N) {
  //verify the solution on the CPU

  //threshold for matching: (0 ok since all vals are small ints)
  float epsilon = 0;  

  for (int i = 0; i < N; i++) {      //for every column...
    for (int j = 0; j < N; j++) {    //for every row in that column
      float tmp = 0;
      for (int k = 0; k < N; k++) {
        tmp += a[i * N + k] * b[k * N + j];
      }

    // Check against the GPU result, throw an error if not equal 
    assert(fabs(c[i * N + j] - tmp) <= epsilon);
    }
  }
}

void check(hipError_t retVal) {
  //takes return value of a CUDA function and checks if it was an error

  if(retVal != hipSuccess) {
    if (retVal==hipErrorInvalidConfiguration)
      printf("Number of Threads per block is not valid");
    fprintf(stderr, "ERROR: %s\n", hipGetErrorString(retVal));
    exit(1);
  }
}

float runTest (float* M, float* N, float* P, float* Md, float* Nd, float* Pd, int size) {
    
  //allocate timers
  hipEvent_t start;
  check(hipEventCreate(&start));
  hipEvent_t stop;
  check(hipEventCreate(&stop));
 
  //start timer
  check(hipEventRecord(start,0));

  //transfer a and b to the GPU
  check(hipMemcpy(Md, M, size, hipMemcpyHostToDevice));
  check(hipMemcpy(Nd, N, size, hipMemcpyHostToDevice));

  //call the kernel
  int gridsize = (WIDTH+TILE_WIDTH-1)/TILE_WIDTH;
  dim3 dimGrid(gridsize, gridsize);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  kernel<<<dimGrid,dimBlock>>>(Md, Nd, Pd, WIDTH);

  //check if kernel encountered an error due to invalid configurations
  hipError_t err = hipGetLastError();
  check(err);

  //transfer result matrix to the host
  check(hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost));

  //stop timer and store it
  check(hipEventRecord(stop,0));
  check(hipEventSynchronize(stop));
  float diff;
  check(hipEventElapsedTime(&diff, start, stop));

  //deallocate timers
  check(hipEventDestroy(start));
  check(hipEventDestroy(stop));

  //print and return time
  printf("Time: %f ms\n", diff);
  return diff;
}

int main() {
  float* M;       //input arrays (on host)
  float* N;
  float* P;       //output array (on host)

  float* Md;      //input arrays (on device)
  float* Nd;
  float* Pd;      //output array (on device)
  
  int size = WIDTH * WIDTH * sizeof(float);  //size of matrix in bytes

  //allocate memory
  M = (float*) malloc(size);
  N = (float*) malloc(size);
  P = (float*) malloc(size);
  check(hipMalloc((void**) &Md, size));
  check(hipMalloc((void**) &Nd, size));
  check(hipMalloc((void**) &Pd, size));

  //fill M and N arrays (all elements <= 2048 so results stay small)
  int cor=0;
  for(int i=0; i < WIDTH * WIDTH; i++){
    M[i] = N[i] = i-cor;
    if(i % 2048 == 0)
      cor=i;
  }

  float total_time = 0;  //accumultate execution times for averaging

  for(int i=0; i < NTESTS; i++)
    total_time += runTest(M, N, P, Md, Nd, Pd, size);

  printf("Avg for %d tests: %f ms and size of matrix %d\n",
	 NTESTS, total_time/(float)NTESTS, WIDTH);
  
  verify_solution(M,N,P,WIDTH);  //verify result 

  //free all memory:
  free(M);
  free(N);
  free(P);
  check(hipFree(Md));
  check(hipFree(Nd));
  check(hipFree(Pd));
}
